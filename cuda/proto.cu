#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>

#include "../src/utils.h"
#include "cudaErrorCheck.h"

#define JITIFY_PRINT_INSTANTIATION 0
#define JITIFY_PRINT_SOURCE 0
#define JITIFY_PRINT_LOG 0
#define JITIFY_PRINT_PTX 0
//#define JITIFY_PRINT_LINKER_LOG 0
#define JITIFY_PRINT_LAUNCH 0
#include "jitify.hpp"

#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wmissing-field-initializers"
#pragma GCC diagnostic ignored "-Wimplicit-fallthrough="
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#pragma GCC diagnostic pop

int main(int argc, char **argv)
{
    if (argc != 3) {
        std::cerr << "USAGE: proto infile.cuh outfile.png\n";
        std::exit(1);
    }
    std::string source_path = argv[1];
    std::string dest_png{argv[2]};

    int magnification = 4;
    int image_width = 720 * magnification;
    int image_height = 720 * magnification;
    int surface_width = image_width;
    int surface_height = image_height;
    assert(surface_width >= image_width);
    assert(surface_height >= image_height);
    int tx = 16;
    int ty = 16;

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image ";
    // std::cerr << "to a " << surface_width << "x" << surface_height << " surface ";
    std::cerr << "in " << tx << "x" << ty << " blocks using the GPU.\n";

    // pan around the image by moving the upper-left corner
    float U0 = 0.0, V0 = 0.0, W0 = 0.0, T0 = 0.0;
    // zoom into the image by adjusting these (> 1 zooms out, < 1 zooms in)
    float dU = 1.0, dV = 1.0;

    float4 image_origin = make_float4(U0, V0, W0, T0);
    float2 image_delta = make_float2(dU, dV);
    std::cerr << "Origin: " << U0 << ", " << V0 << ", " << W0 << ", " << T0 << ".\n";
    std::cerr << " Delta: " << dU << ", " << dV << ".\n";

    int surface_pixels = surface_width * surface_height;
    size_t fb_bytes = 4 * surface_pixels * sizeof(uint8_t);

    // allocate Frame Buffer (FB) on the GPU
    uint8_t *fb;
    cudaErrChk(hipMallocManaged((void **)&fb, fb_bytes));

    // nvrt compile code.  Requires a module name as first line.
    // Use JITIFY_OPTIONS="-I/path/to/this/tweegeemeetranspiler" if not running
    // from above the cuda directory.
    std::string proto_src = "proto\n"
                            "#include \"cuda/clisk.cuh\"\n" +
                            slurp(source_path);
    static jitify::JitCache kernel_cache;
    jitify::Program program = kernel_cache.program(proto_src);

    clock_t start, stop;
    start = clock();
    // Render to our FB
    dim3 blocks(image_width / tx + 1, image_height / ty + 1);
    dim3 threads(tx, ty);
    cuErrChk(program.kernel("render_rgba")
                 .instantiate()
                 .configure(blocks, threads)
                 .launch(fb, image_origin, image_delta, image_width, image_height, surface_width));
    cudaErrChk(hipGetLastError());
    cudaErrChk(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // bring FB back to the CPU memory (no error check since this is just for performance and can fail on some systems)
    hipMemPrefetchAsync(fb, fb_bytes, hipCpuDeviceId);

    // Output FB as PNG Image
    std::cerr << "writing to " << dest_png << "...\n";
    stbi_write_png(dest_png.c_str(), image_width, image_height, 4, (const void *)fb,
                   surface_width * 4 * sizeof(uint8_t));

    cudaErrChk(hipFree(fb));
}
